#include "hip/hip_runtime.h"
#include "./Measurements.cu"
#include "./Pragmas.cu"

__global__ void strategy1DNestedKernel(const int *input, int* output)
{
	__shared__ int sdata[THREADBLOCKSIZE];
	const unsigned int tx = threadIdx.x;
	const unsigned int bx = blockIdx.x; 
	const size_t offset = bx * blockDim.x + tx;
	
	// Read data
	sdata[tx] = offset < INPUTSIZE ? input[offset] : 0;
	__syncthreads();
	
	// Folding
	#if THREADBLOCKSIZE >= 1024
	PART_NESTED_REDUCTION_X(sdata, tx, 512)
	#endif
	#if THREADBLOCKSIZE >= 512
	PART_NESTED_REDUCTION_X(sdata, tx, 256)
	#endif
	#if THREADBLOCKSIZE >= 256
	PART_NESTED_REDUCTION_X(sdata, tx, 128)
	#endif
	#if THREADBLOCKSIZE >= 128
	PART_NESTED_REDUCTION_X(sdata, tx, 64)
	#endif
	#if THREADBLOCKSIZE >= 64
	PART_NESTED_REDUCTION_X(sdata, tx, 32)
	#endif
	#if THREADBLOCKSIZE >= 32
	PART_NESTED_REDUCTION_X(sdata, tx, 16)
	#endif
	#if THREADBLOCKSIZE >= 16
	PART_NESTED_REDUCTION_X(sdata, tx, 8)
	#endif
	#if THREADBLOCKSIZE >= 8
	PART_NESTED_REDUCTION_X(sdata, tx, 4)
	#endif
	#if THREADBLOCKSIZE >= 4
	PART_NESTED_REDUCTION_X(sdata, tx, 2)
	#endif
	#if THREADBLOCKSIZE >= 2
	if(tx == 0)
		output[bx] = sdata[0] + sdata[1];
	#endif
	
	#if THREADBLOCKSIZE >= 4
	}
	#if THREADBLOCKSIZE >= 8
	}
	#if THREADBLOCKSIZE >= 16
	}
	#if THREADBLOCKSIZE >= 32
	}
	#if THREADBLOCKSIZE >= 64
	}
	#if THREADBLOCKSIZE >= 128
	}
	#if THREADBLOCKSIZE >= 256
	}
	#if THREADBLOCKSIZE >= 512
	}
	#if THREADBLOCKSIZE >= 1024
	}
	#endif
	#endif
	#endif
	#endif
	#endif
	#endif
	#endif
	#endif
	#endif
}

void strategy1DNested(Measurements* measurements)
{
    const size_t inputByteSize = INPUTSIZE * sizeof(int);
	const size_t outputByteSize = GRIDSIZE * sizeof(int);
	int *h_input, *d_input, *h_output, *d_output;
	int res = 0, trueRes = 0;
	
	// Initialise input arrays with 1s
	h_input = (int*)mallocMeasurements(inputByteSize, measurements);
	for(size_t i = 0; i < INPUTSIZE; ++i) {
		h_input[i] = i; 
		trueRes += i; 
	}
	
	hipMalloc(&d_input, INPUTSIZE * sizeof(int));
	hipMemcpy(d_input, h_input, inputByteSize, hipMemcpyHostToDevice);
	
	// Initialise output arrays
	h_output = (int*)mallocMeasurements(outputByteSize, measurements);
	hipMalloc(&d_output, outputByteSize);
	
	// Fold operation
	for(int i = 0; i < ITERATIONS; ++i) {
		startGpuTimer(measurements);
		strategy1DNestedKernel<<<GRIDSIZE, THREADBLOCKSIZE>>>(d_input, d_output);
		recordTime(measurements, measurements->gpuFoldTime, stopGpuTimer(measurements));
		hipDeviceSynchronize();
		measurements->iterations++;
	}
	hipMemcpy(h_output, d_output, outputByteSize, hipMemcpyDeviceToHost);
	
	// Linear fold on CPU
	for(size_t i = 0; i < GRIDSIZE; ++i)
		res += h_output[i];
	
	// Update final measurements
	measurements->success = measurements->success && res == trueRes;
	checkGpuErrors(measurements);
	
	// Free everything
	free(h_input);
	free(h_output);
	hipFree(d_input);
	hipFree(d_output);
}

#ifndef HASMAIN
#define HASMAIN
int main(void)
{
	char foldName[] = "Strategy_1DNested";
	Measurements* measurements = measurements_new((char*)&foldName);
	strategy1DNested(measurements);
	printMeasurements(measurements);
	
	free(measurements);
	
	return 0;
}
#endif
