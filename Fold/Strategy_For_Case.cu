#include "hip/hip_runtime.h"
#include "./Measurements.cu"

__global__ void strategyForKernel(const int *input, int* output)
{
	__shared__ int sdata[THREADBLOCKSIZE];
	const unsigned int tx = threadIdx.x;
	const unsigned int bx = blockIdx.x; 
	const size_t offset = bx * blockDim.x + tx;
	
	// Read data
	sdata[tx] = offset < INPUTSIZE ? input[offset] : 0;
	__syncthreads();
	
	// Folding
	if(tx == 0) {
		for(int i = 1; i < THREADBLOCKSIZE; ++i)
			sdata[0] += sdata[i];
		output[bx] = sdata[0];
	}
}

void strategyFor(Measurements* measurements)
{
    const size_t inputByteSize = INPUTSIZE * sizeof(int);
	const size_t outputByteSize = GRIDSIZE * sizeof(int);
	int *h_input, *d_input, *h_output, *d_output;
	int res = 0, trueRes = 0;
	
	// Initialise input arrays with 1s
	h_input = (int*)mallocMeasurements(inputByteSize, measurements);
	for(size_t i = 0; i < INPUTSIZE; ++i) {
		h_input[i] = i; 
		trueRes += i; 
	}
	
	hipMalloc(&d_input, INPUTSIZE * sizeof(int));
	hipMemcpy(d_input, h_input, inputByteSize, hipMemcpyHostToDevice);
	
	// Initialise output arrays
	h_output = (int*)mallocMeasurements(outputByteSize, measurements);
	hipMalloc(&d_output, outputByteSize);
	
	// Fold operation
	for(int i = 0; i < ITERATIONS; ++i) {
		startGpuTimer(measurements);
		strategyForKernel<<<GRIDSIZE, THREADBLOCKSIZE>>>(d_input, d_output);
		recordTime(measurements, measurements->gpuFoldTime, stopGpuTimer(measurements));
		hipDeviceSynchronize();
		measurements->iterations++;
	}
	hipMemcpy(h_output, d_output, outputByteSize, hipMemcpyDeviceToHost);
	
	// Linear fold on CPU
	for(size_t i = 0; i < GRIDSIZE; ++i)
		res += h_output[i];
	
	// Update final measurements
	measurements->success = measurements->success && res == trueRes;
	checkGpuErrors(measurements);
	
	// Free everything
	free(h_input);
	free(h_output);
	hipFree(d_input);
	hipFree(d_output);
}

#ifndef HASMAIN
#define HASMAIN
int main(void)
{
	char foldName[] = "Strategy_For";
	Measurements* measurements = measurements_new((char*)&foldName);
	strategyFor(measurements);
	printMeasurements(measurements);
	
	free(measurements);
	
	return 0;
}
#endif
