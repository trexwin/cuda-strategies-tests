#include "hip/hip_runtime.h"
#include "./Measurements.cu"
#include "./Pragmas.cu"

__global__ void strategy2DUnrolledKernel(const int *input, int* output)
{
	__shared__ int sdata[THREADBLOCKSIZE/32][32];
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;
	const unsigned int bx = blockIdx.x;
	const size_t offset = bx * THREADBLOCKSIZE + (ty * 32 + tx);
	
	// Read data
	sdata[ty][tx] = offset < INPUTSIZE ? input[offset] : 0;
	__syncthreads();
	
	// Folding
	#if THREADBLOCKSIZE >= 1024
	BASIC_REDUCTION_Y(sdata, tx, ty, 16)
	#endif
	#if THREADBLOCKSIZE >= 512
	BASIC_REDUCTION_Y(sdata, tx, ty, 8)
	#endif
	#if THREADBLOCKSIZE >= 256
	BASIC_REDUCTION_Y(sdata, tx, ty, 4)
	#endif
	#if THREADBLOCKSIZE >= 128
	BASIC_REDUCTION_Y(sdata, tx, ty, 2)
	#endif
	#if THREADBLOCKSIZE >= 64
	BASIC_REDUCTION_Y(sdata, tx, ty, 1)
	#endif
	
	BASIC_REDUCTION_X(sdata[ty], tx, 16)
	BASIC_REDUCTION_X(sdata[ty], tx, 8)
	BASIC_REDUCTION_X(sdata[ty], tx, 4)
	BASIC_REDUCTION_X(sdata[ty], tx, 2)
	
	if(tx == 0 && ty == 0)
		output[bx] = sdata[0][tx] + sdata[0][tx + 1];
	
}

void strategy2DUnrolled(Measurements* measurements)
{
    const size_t inputByteSize = INPUTSIZE * sizeof(int);
	const size_t outputByteSize = GRIDSIZE * sizeof(int);
	int *h_input, *d_input, *h_output, *d_output;
	int res = 0, trueRes = 0;
	
	// Initialise input arrays with 1s
	h_input = (int*)mallocMeasurements(inputByteSize, measurements);
	for(size_t i = 0; i < INPUTSIZE; ++i) {
		h_input[i] = i; 
		trueRes += i; 
	}
	
	hipMalloc(&d_input, INPUTSIZE * sizeof(int));
	hipMemcpy(d_input, h_input, inputByteSize, hipMemcpyHostToDevice);
	
	// Initialise output arrays
	h_output = (int*)mallocMeasurements(outputByteSize, measurements);
	hipMalloc(&d_output, outputByteSize);
	
	// Fold operation
	for(int i = 0; i < ITERATIONS; ++i) {
		startGpuTimer(measurements);
		strategy2DUnrolledKernel<<<GRIDSIZE, dim3(32, THREADBLOCKSIZE/32)>>>(d_input, d_output);
		recordTime(measurements, measurements->gpuFoldTime, stopGpuTimer(measurements));
		hipDeviceSynchronize();
		measurements->iterations++;
	}
	hipMemcpy(h_output, d_output, outputByteSize, hipMemcpyDeviceToHost);
	
	// Linear fold on CPU
	for(size_t i = 0; i < GRIDSIZE; ++i)
		res += h_output[i];
	
	// Update final measurements
	measurements->success = measurements->success && res == trueRes;
	checkGpuErrors(measurements);
	
	// Free everything
	free(h_input);
	free(h_output);
	hipFree(d_input);
	hipFree(d_output);
}

#ifndef HASMAIN
#define HASMAIN
int main(void)
{
	char foldName[] = "Strategy_2DUnrolled";
	Measurements* measurements = measurements_new((char*)&foldName);
	strategy2DUnrolled(measurements);
	printMeasurements(measurements);
	
	free(measurements);
	
	return 0;
}
#endif
