#include "hip/hip_runtime.h"
#include "./Measurements.cu"
#include "./Pragmas.cu"

__global__ void baseCaseKernel(int *input, int* output)
{
	__shared__ int sdata[THREADBLOCKSIZE];
	const unsigned int tx = threadIdx.x;
	const unsigned int bx = blockIdx.x; 
	const size_t offset = bx * blockDim.x + tx;
	
	// Read data
	sdata[tx] = offset < INPUTSIZE ? input[offset] : 0;
	__syncthreads();
	
	// Folding
	FOR_BASIC_REDUCTION_X(sdata, tx, THREADBLOCKSIZE/2, 0)
	
	if(tx == 0)
		output[bx] = sdata[0];
}

void baseCase(Measurements* measurements)
{
	const size_t inputByteSize = INPUTSIZE * sizeof(int);
	const size_t outputByteSize = GRIDSIZE * sizeof(int);
	int *h_input, *d_input, *h_output, *d_output;
	int res = 0, trueRes = 0;
	
	// Initialise input arrays with 1s
	h_input = (int*)mallocMeasurements(inputByteSize, measurements);
	for(size_t i = 0; i < INPUTSIZE; ++i) {
		h_input[i] = i; 
		trueRes += i; 
	}
	
	hipMalloc(&d_input, INPUTSIZE * sizeof(int));
	hipMemcpy(d_input, h_input, inputByteSize, hipMemcpyHostToDevice);
	
	// Initialise output arrays
	h_output = (int*)mallocMeasurements(outputByteSize, measurements);
	hipMalloc(&d_output, outputByteSize);
	
	// Fold operation
	for(int i = 0; i < ITERATIONS; ++i) {
		startGpuTimer(measurements);
		baseCaseKernel<<<GRIDSIZE, THREADBLOCKSIZE>>>(d_input, d_output);
		recordTime(measurements, measurements->gpuFoldTime, stopGpuTimer(measurements));
		hipDeviceSynchronize();
		measurements->iterations++;
	}
	hipMemcpy(h_output, d_output, outputByteSize, hipMemcpyDeviceToHost);
	
	// Linear fold on CPU
	for(size_t i = 0; i < GRIDSIZE; ++i)
		res += h_output[i];
	
	// Update final measurements
	measurements->success = measurements->success && res == trueRes;
	checkGpuErrors(measurements);
	
	// Free everything
	free(h_input);
	free(h_output);
	hipFree(d_input);
	hipFree(d_output);
}

#ifndef HASMAIN
#define HASMAIN
int main(void)
{
	char foldName[] = "Base";
	Measurements* measurements = measurements_new((char*)&foldName);
	baseCase(measurements);
	printMeasurements(measurements);
	
	free(measurements);
	
	return 0;
}
#endif
